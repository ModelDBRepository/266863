
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <math.h>


#define I	7
#define dt	0.01
#define a    0.02
#define b    0.2
#define c  -65.
#define d    6.


// Set GPU parallelization 
#define BLOCKS  4
#define THREADS 256

// Set simulation time
#define TIME_ITERATIONS 6000000l


__global__
void run(float *v, float *u)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	for(unsigned long t   = 0; t<TIME_ITERATIONS;    ++t){
		v[i] += dt*(0.04f*v[i]*v[i]+5.f*v[i]+140.f-u[i]+I);
		u[i] += dt*a*(b*v[i]-u[i]);
		if (v[i] > 30.f){
			v[i] = c;
			u[i] = u[i] + d;
		}
	}
	
}

int main(void)
{
	int N = BLOCKS*THREADS;
	float *v, *u;

	// Allocate Unified Memory – accessible from CPU or GPU
	hipMallocManaged(&v, N*sizeof(float));
	hipMallocManaged(&u, N*sizeof(float));
	

	// initialize arrays on the host
	for (int i = 0; i < N; i++) {
		v[i] = -70.f;
		u[i] = -14.f;
	}

	// Run kernel on the GPU
	run<<<BLOCKS, THREADS>>>(v, u);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();
	
	//check for errors
	hipError_t e = hipGetLastError();
	if(e){
		printf("ERROR (%d): %s\n",e,hipGetErrorString(e));
	}

	// Free memory
	hipFree(v);
	hipFree(u);

	return 0;
}
