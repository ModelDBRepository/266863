
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <math.h>


#define I	7
#define gna	35.
#define gk	9.
#define gl	0.1
#define ena	55.
#define ek	(-90.)
#define el	(-65.)
#define dt	0.01

// Set GPU parallelization 
#define BLOCKS  4
#define THREADS 256

// Set simulation time
#define TIME_ITERATIONS 6000000l

__global__
void run(float *v, float *h, float *n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	float minf, hinf, htau, ninf, ntau, a, b, vm, dv=(ena-ek)/201., ras;
	float tbl[201][6];
	int  vi;
	for(unsigned long t=0; t<20l; t++){
		tbl[t][0] = vm = ek+dv*t;
		a = 0.1*(vm+35.)/(1.0-exp(-(vm+35.)/10.)) ;
		b = 4.0*exp(-(vm+60.)/18.);
		tbl[t][1] = a/(a+b);
		a = 0.07*exp(-(vm+58.)/20.);
		b = 1.0/(1.0+exp(-(vm+28.)/10.));
		tbl[t][2] =  a/(a+b);
		tbl[t][3] = 1./(a+b);
		a = 0.01*(vm+34.)/(1.0-exp(-(vm+34.)/10.));
		b = 0.125*exp(-(vm+44.)/80.);
		tbl[t][4] =  a/(a+b);
		tbl[t][5] = 1./(a+b);
	}
	for(unsigned long t   = 0; t<TIME_ITERATIONS;    ++t){
		vi = (int)floor((v[i]-ek)/dv);
		ras   = (v[i] - tbl[vi][0])/dv;
		minf = tbl[vi][1] + (tbl[vi+1][1] - tbl[vi][1])*ras;
		hinf = tbl[vi][2] + (tbl[vi+1][2] - tbl[vi][2])*ras;
		htau = tbl[vi][3] + (tbl[vi+1][3] - tbl[vi][3])*ras;
		ninf = tbl[vi][4] + (tbl[vi+1][4] - tbl[vi][4])*ras;
		ntau = tbl[vi][5] + (tbl[vi+1][5] - tbl[vi][5])*ras;
		
		n[i] += dt*(ninf - n[i])/ntau;
		h[i] += dt*(hinf - h[i])/htau;
		v[i] += dt*(-gna*minf*minf*minf*h[i]*(v[i]-ena)-gk*n[i]*n[i]*n[i]*n[i]*(v[i]-ek)-gl*(v[i]-el)+I);
	}
	
}

int main(void)
{
	int N = BLOCKS*THREADS;
	float *v, *h, *n;

	// Allocate Unified Memory – accessible from CPU or GPU
	hipMallocManaged(&v, N*sizeof(float));
	hipMallocManaged(&h, N*sizeof(float));
	hipMallocManaged(&n, N*sizeof(float));

	// initialize arrays on the host
	for (int i = 0; i < N; i++) {
		v[i] = -63.f;
		h[i] = n[i] = 0.f;
	}

	// Run kernel on the GPU
	run<<<BLOCKS, THREADS>>>(v, h, n);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();
	
	//check for errors
	hipError_t e = hipGetLastError();
	if(e){
		printf("ERROR (%d): %s\n",e,hipGetErrorString(e));
	}

	// Free memory
	hipFree(v);
	hipFree(h);
	hipFree(n);

	return 0;
}
