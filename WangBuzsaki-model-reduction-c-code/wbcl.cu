
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <math.h>


#define I	7
#define gna	35.
#define gk	9.
#define gl	0.1
#define ena	55.
#define ek	(-90.)
#define el	(-65.)
#define dt	0.01

// Set GPU parallelization 
#define BLOCKS  4
#define THREADS 256

// Set simulation time
#define TIME_ITERATIONS 6000000l


__global__
void run(float *v, float *h, float *n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	float minf, ninf, hinf, ntau, htau, a, b;
	for(unsigned long t   = 0; t<TIME_ITERATIONS;    ++t){
		a = 0.1*(v[i]+35.)/(1.0-exp(-(v[i]+35.)/10.)) ;
		b = 4.0*exp(-(v[i]+60.)/18.);
		minf = a/(a+b);
		
		a = 0.01*(v[i]+34.)/(1.0-exp(-(v[i]+34.)/10.));
		b = 0.125*exp(-(v[i]+44.)/80.);
		ninf =  a/(a+b);
		ntau = 1./(a+b);
		
		a = 0.07*exp(-(v[i]+58.)/20.);
		b = 1.0/(1.0+exp(-(v[i]+28.)/10.));
		hinf =  a/(a+b);
		htau = 1./(a+b);

		n[i] += dt*(ninf - n[i])/ntau;
		h[i] += dt*(hinf - h[i])/htau;
		v[i] += dt*(-gna*minf*minf*minf*h[i]*(v[i]-ena)-gk*n[i]*n[i]*n[i]*n[i]*(v[i]-ek)-gl*(v[i]-el)+I);
	}
	
}

int main(void)
{
	int N = BLOCKS*THREADS;
	float *v, *h, *n;

	// Allocate Unified Memory – accessible from CPU or GPU
	hipMallocManaged(&v, N*sizeof(float));
	hipMallocManaged(&h, N*sizeof(float));
	hipMallocManaged(&n, N*sizeof(float));

	// initialize arrays on the host
	for (int i = 0; i < N; i++) {
		v[i] = -63.f;
		h[i] = n[i] = 0.f;
	}

	// Run kernel on the GPU
	run<<<BLOCKS, THREADS>>>(v, h, n);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();
	
	//check for errors
	hipError_t e = hipGetLastError();
	if(e){
		printf("ERROR (%d): %s\n",e,hipGetErrorString(e));
	}

	// Free memory
	hipFree(v);
	hipFree(h);
	hipFree(n);

	return 0;
}
