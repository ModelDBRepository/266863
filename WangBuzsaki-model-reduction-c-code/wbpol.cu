
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <math.h>
// Kernel function to add the elements of two arrays

#define I	7
#define gna	35.
#define gk	9.
#define gl	0.1
#define ena	55.
#define ek	(-90.)
#define el	(-65.)
#define dt	0.01

// Set GPU parallelization 
#define BLOCKS  4
#define THREADS 256

// Set simulation time
#define TIME_ITERATIONS 6000000l

__global__
void run(float *v, float *h, float *n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	float minf, ninf, hinf, ntau, htau;
	for(unsigned long t   = 0; t<TIME_ITERATIONS;    ++t){
		minf =  0.953881978928+v[i]*(0.00764436291528+v[i]*(-0.000176887049516+v[i]*(-1.95108741101e-07+v[i]*(1.66385198983e-08+v[i]*(1.67105570231e-11)))));
		
		ninf =  0.824494447868+v[i]*(0.00719826506481+v[i]*(-0.000110730240325+v[i]*(-1.53119478562e-07+v[i]*(9.39864044908e-09+v[i]*(3.03369442673e-12)))));
		ntau =  0.492403741388+v[i]*(-0.012052097973+v[i]*(0.00010471373932+v[i]*(1.69326656183e-06+v[i]*(-1.44449337737e-08+v[i]*(-1.12462063995e-10)))));
		
		hinf =  -0.0310904478177+v[i]*(0.00192650413623+v[i]*(8.72227960325e-05+v[i]*(-2.94688600355e-06+v[i]*(6.87296459746e-11+v[i]*(2.70678460466e-10)))));
		htau = 	 0.162350260789+v[i]*(-0.00768669406983+v[i]*(0.000298711718949+v[i]*(-8.22822927459e-07+v[i]*(-3.45692137243e-08+v[i]*(1.43004040707e-10)))));

		n[i] += dt*(ninf - n[i])/ntau;
		h[i] += dt*(hinf - h[i])/htau;
		v[i] += dt*(-gna*minf*minf*minf*h[i]*(v[i]-ena)-gk*n[i]*n[i]*n[i]*n[i]*(v[i]-ek)-gl*(v[i]-el)+I);
	}
}

int main(void)
{
	int N = BLOCKS*THREADS;
	float *v, *h, *n;

	// Allocate Unified Memory – accessible from CPU or GPU
	hipMallocManaged(&v, N*sizeof(float));
	hipMallocManaged(&h, N*sizeof(float));
	hipMallocManaged(&n, N*sizeof(float));

	// initialize arrays on the host
	for (int i = 0; i < N; i++) {
		v[i] = -63.f;
		h[i] = n[i] = 0.f;
	}

	// Run kernel on the GPU
	run<<<BLOCKS, THREADS>>>(v, h, n);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	//check for errors
	hipError_t e = hipGetLastError();
	if(e){
		printf("ERROR (%d): %s\n",e,hipGetErrorString(e));
	}

	// Free memory
	hipFree(v);
	hipFree(h);
	hipFree(n);

	return 0;
}
