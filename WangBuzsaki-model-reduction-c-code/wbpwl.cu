
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <math.h>
// Kernel function to add the elements of two arrays

#define I	7
#define gna	35.
#define gk	9.
#define gl	0.1
#define ena	55.
#define ek	(-90.)
#define el	(-65.)
#define dt	0.01

// Set GPU parallelization 
#define BLOCKS  4
#define THREADS 256

// Set simulation time
#define TIME_ITERATIONS 6000000l

__global__
void run(float *v, float *h, float *n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	float minf, ninf, hinf, ntau, htau;
	for(unsigned long t   = 0; t<TIME_ITERATIONS;    ++t){
		minf =  (0.0313077505826+0.000327941702344*v[i])+\
				((v[i]>-63.8824871877)?(0.984216321118+0.0154066687827*v[i]):0.)+\
				((v[i]>-6.79083019467)?(-0.0960057625861+-0.0141375590074*v[i]):0.);
		
		ninf =  (0.0964837408311+0.00095221890206*v[i])+\
				((v[i]>-71.0303053169)?(0.743014681244+0.0104605305852*v[i]):0.)+\
				((v[i]>-4.02151450044)?(-0.0343121915059+-0.00853215660471*v[i]):0.);
		ntau =  (3.89986986823+0.0365576715394*v[i])+\
				((v[i]>-60.3962620374)?(-4.73857532036+-0.0784580893008*v[i]):0.)+\
				((v[i]>-26.7339539639)?(1.09061524502+0.0407951344007*v[i]):0.);
		
		hinf =  (0.961765215359+-0.000371676284993*v[i])+\
				((v[i]>-81.574862488)?(-1.32606283442+-0.0162557777479*v[i]):0.)+\
				((v[i]>-25.0499306576)?(0.397064011884+0.0158509026357*v[i]):0.);
		htau = 	(1.78046831377+0.00985969575028*v[i])+\
				((v[i]>-57.5852330206)?(-1.29845989067+-0.0225484872173*v[i]):0.)+\
				((v[i]>5.61104313787)?(-0.0499171171021+0.00889622764887*v[i]):0.);

		n[i] += dt*(ninf - n[i])/ntau;
		h[i] += dt*(hinf - h[i])/htau;
		v[i] += dt*(-gna*minf*minf*minf*h[i]*(v[i]-ena)-gk*n[i]*n[i]*n[i]*n[i]*(v[i]-ek)-gl*(v[i]-el)+I);
		//printf("%g %g\n",dt*t,v);
	}
}

int main(void)
{
	int N = BLOCKS*THREADS;
	float *v, *h, *n;

	// Allocate Unified Memory – accessible from CPU or GPU
	hipMallocManaged(&v, N*sizeof(float));
	hipMallocManaged(&h, N*sizeof(float));
	hipMallocManaged(&n, N*sizeof(float));

	// initialize arrays on the host
	for (int i = 0; i < N; i++) {
		v[i] = -63.f;
		h[i] = n[i] = 0.f;
	}

	// Run kernel on the GPU
	run<<<BLOCKS, THREADS>>>(v, h, n);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	//check for errors
	hipError_t e = hipGetLastError();
	if(e){
		printf("ERROR (%d): %s\n",e,hipGetErrorString(e));
	}

	// Free memory
	hipFree(v);
	hipFree(h);
	hipFree(n);

	return 0;
}
