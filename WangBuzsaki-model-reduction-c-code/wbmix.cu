
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <math.h>


#define I	7
#define gna	35.
#define gk	9.
#define gl	0.1
#define ena	55.
#define ek	(-90.)
#define el	(-65.)
#define dt	0.01

// Set GPU parallelization 
#define BLOCKS  4
#define THREADS 256

// Set simulation time
#define TIME_ITERATIONS 6000000l

__global__
void run(float *v, float *n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	float ninf, ntau, vinf, vtau;
	for(unsigned long t   = 0; t<TIME_ITERATIONS;    ++t){
		vinf = (v[i]+61.);
		vinf = (I+5e-4*vinf*vinf*(55.-v[i])-0.13)* ((v[i]<-61.)?(1. + 0.034*(61.+v[i])):1.);
		vtau =(v[i]>-38.)? 0.2:((-38.-v[i])*0.05+0.2);

		ninf =  (0.0964837408311+0.00095221890206*v[i])+\
				((v[i]>-71.0303053169)?(0.743014681244+0.0104605305852*v[i]):0.)+\
				((v[i]>-4.02151450044)?(-0.0343121915059+-0.00853215660471*v[i]):0.);
		ntau =  (3.89986986823+0.0365576715394*v[i])+\
				((v[i]>-60.3962620374)?(-4.73857532036+-0.0784580893008*v[i]):0.)+\
				((v[i]>-26.7339539639)?(1.09061524502+0.0407951344007*v[i]):0.);
		

		n[i] += dt*(ninf - n[i])/ntau;
		v[i] += dt*(vinf-gk*n[i]*n[i]*n[i]*n[i]*(v[i]-ek))/vtau;
	}
	
}

int main(void)
{
	int N = BLOCKS*THREADS;
	float *v, *n;

	// Allocate Unified Memory – accessible from CPU or GPU
	hipMallocManaged(&v, N*sizeof(float));
	hipMallocManaged(&n, N*sizeof(float));

	// initialize arrays on the host
	for (int i = 0; i < N; i++) {
		v[i] = -63.f;
		n[i] = 0.f;
	}

	// Run kernel on the GPU
	run<<<BLOCKS, THREADS>>>(v, n);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();
	
	//check for errors
	hipError_t e = hipGetLastError();
	if(e){
		printf("ERROR (%d): %s\n",e,hipGetErrorString(e));
	}

	// Free memory
	hipFree(v);
	hipFree(n);

	return 0;
}
